#include "hip/hip_runtime.h"

#include "kernel.h"



__global__ void circularConvKernel(const float* x1, const float* x2, const int* size, float* result)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < *size) {
        float val = 0;

        for (int i = 0; i < *size; i++) {
            int index = (id - i + *size) % *size;
            val += x1[i] * x2[index];
        }
        
        result[id] = val;
    }
}

void gpuRev(const float* dryBuffer,const  float* irBuffer, const int irBufferSize, int blocks, int threads, float* out)
 {
 
 	 


	float* d_wetBuffer;
	hipMalloc((void**)&d_wetBuffer, (irBufferSize) * sizeof(float));

	float* d_dryBufferC;
	hipMalloc((void**)&d_dryBufferC, (irBufferSize) * sizeof(float));
	
	float* d_impBufferC;
	hipMalloc((void**)&d_impBufferC, (irBufferSize) * sizeof(float));
	
	int* d_size;
	hipMalloc((void**)&d_size, sizeof(int));
	
	//copy to gpu
	hipMemcpy(d_size, &irBufferSize, sizeof(int), hipMemcpyHostToDevice);
 	hipMemcpy(d_dryBufferC, dryBuffer, (irBufferSize) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_impBufferC, irBuffer, (irBufferSize) * sizeof(float), hipMemcpyHostToDevice);
 	
 	//perform
	 circularConvKernel<<<blocks, threads>>>(d_dryBufferC,d_impBufferC, d_size, d_wetBuffer);
	 hipDeviceSynchronize();
	 // Wait for GPU to finish before accessing on host


	 
	
	 hipMemcpy(out, d_wetBuffer, (irBufferSize) * sizeof(float), hipMemcpyDeviceToHost);


	  
	 
	// Free device and host memory
    	 
   	 hipFree(d_dryBufferC);
   	 hipFree(d_impBufferC);
   	 hipFree(d_wetBuffer);
   	 hipFree(d_size);
    
}


