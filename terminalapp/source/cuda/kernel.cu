#include "hip/hip_runtime.h"

#include "kernel.h"



__global__ void circularConvKernel( float* x1,  float* x2, const int* size, float* result, const  int* channels)
{
   	const int c = blockIdx.x * blockDim.x + threadIdx.x; // samples
    	int ch = 0;

    	if (c < *size * *channels) {
       		int offset = ch  * (*size); // Offset to the start of the current channel

        	float val = 0;

        	for (int i = 0; i < *size; i++) {
            		int index = (c - i + *size) % *size;
            		val += x1[offset + i] * x2[offset + index];
        }

        	result[offset + c] = val;
        	 ch++;
    }
   
}


__global__ void cu_mult(float* x1, float* scale, const  int* channels, const  int* size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    for(unsigned int ch = 0; ch < *channels; ch++)
    {
        int index = ch * (*size) + id;
        x1[index] *= *scale;
    }
}
void gpuRev(float* dryBuffer,float* irBuffer, const int bufferSize, float* out, const unsigned int channels)
{

	int threads = 1024;

	int blocks = ((int)(bufferSize * channels) / threads) + 1;


    	float* d_wetBuffer;
    	float* d_dryBufferC;
    	float* d_impBufferC;
    	
    	int* d_size;
    	hipMalloc((void**)&d_size, sizeof(int));

    	int* d_channels;
    	hipMalloc((void**)&d_channels, sizeof( int));

    	float* d_scale;
    	float h_scale = 0.15;
    	hipMalloc((void**)&d_scale, sizeof(float));
    	hipMemcpy(d_scale, &h_scale, sizeof(float), hipMemcpyHostToDevice);

    // Copy to GPU
    	hipMemcpy(d_size, &bufferSize, sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(d_channels, &channels, sizeof( int), hipMemcpyHostToDevice);


	hipMalloc( (void**)&d_wetBuffer ,bufferSize * channels*sizeof(float) );
	hipMalloc( (void**)&d_dryBufferC ,bufferSize * channels*sizeof(float) );
	hipMalloc( (void**)&d_impBufferC , bufferSize * channels*sizeof(float) );
	
	hipMemcpy(d_dryBufferC,dryBuffer,bufferSize * channels*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_impBufferC,irBuffer,bufferSize * channels*sizeof(float),hipMemcpyHostToDevice);
	 

	// Perform circular convolution
	circularConvKernel<<<blocks, threads>>>(d_dryBufferC, d_impBufferC, d_size, d_wetBuffer, d_channels);
	hipDeviceSynchronize();

	// Perform multiplication/normalisation
	//cu_mult<<<gridSize_mult, blockSize_mult>>>(d_wetBuffer, d_scale, d_channels, d_size);
	//hipDeviceSynchronize();

    // Copy result back to host
    	hipMemcpy(out,d_wetBuffer,bufferSize * channels*sizeof(float), hipMemcpyDeviceToHost);

     

    hipFree(d_size);
    hipFree(d_channels);
    hipFree(d_scale);
    hipFree(d_wetBuffer);
    hipFree(d_dryBufferC);
    hipFree(d_impBufferC);
}
